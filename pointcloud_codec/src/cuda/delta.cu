// cuda/delta.cu
#include <hip/hip_runtime.h>
#include <cstdint>

extern "C" {

__global__ void pcc_delta_u32_triplets_kernel(const std::uint32_t* __restrict__ in,
                                              std::uint32_t* __restrict__ out,
                                              int N) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    if (i == 0) {
        out[0] = in[0];
        out[1] = in[1];
        out[2] = in[2];
        return;
    }
    const int k = 3*i;
    const int km = 3*(i-1);
    out[k+0] = in[k+0] - in[km+0];
    out[k+1] = in[k+1] - in[km+1];
    out[k+2] = in[k+2] - in[km+2];
}

} 
