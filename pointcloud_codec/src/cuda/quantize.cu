#include "hip/hip_runtime.h"
// cuda/quantize.cu
#include <hip/hip_runtime.h>
#include <cstdint>
#include "pcc/config.hpp"

extern "C" {

__global__ void pcc_quantize_xyz_kernel(const pcc::float3* __restrict__ in,
                                        std::uint32_t* __restrict__ out, // 3*N
                                        int N,
                                        float3 aabb_min,
                                        float3 aabb_max,
                                        int bits)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    const float L = (1u << bits) - 1u;
    const float rx = fmaxf(aabb_max.x - aabb_min.x, 1e-20f);
    const float ry = fmaxf(aabb_max.y - aabb_min.y, 1e-20f);
    const float rz = fmaxf(aabb_max.z - aabb_min.z, 1e-20f);

    const float3 p = make_float3(in[i].x, in[i].y, in[i].z);

    const float qx = roundf(fminf(fmaxf((p.x - aabb_min.x)/rx, 0.f), 1.f) * L);
    const float qy = roundf(fminf(fmaxf((p.y - aabb_min.y)/ry, 0.f), 1.f) * L);
    const float qz = roundf(fminf(fmaxf((p.z - aabb_min.z)/rz, 0.f), 1.f) * L);

    out[3*i + 0] = static_cast<std::uint32_t>(qx);
    out[3*i + 1] = static_cast<std::uint32_t>(qy);
    out[3*i + 2] = static_cast<std::uint32_t>(qz);
}

} 
